#include <hip/hip_runtime.h>
#include <iostream>

int main() {
    int device_count;
    hipGetDeviceCount(&device_count);

    if (device_count == 0) {
        std::cout << "No CUDA devices found." << std::endl;
        return 1;
    }

    for (int i = 0; i < device_count; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        std::cout << "GPU " << i << ": " << prop.name << std::endl;
    }
    return 0;
}
