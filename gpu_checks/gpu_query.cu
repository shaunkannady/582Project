#include <hip/hip_runtime.h>
#include <iostream>

int main() {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);  // 0 = device ID
    std::cout << "Max threads per block: " << prop.maxThreadsPerBlock << std::endl;
    std::cout << "Max threads dim: [" 
              << prop.maxThreadsDim[0] << ", "
              << prop.maxThreadsDim[1] << ", "
              << prop.maxThreadsDim[2] << "]" << std::endl;
    std::cout << "Max grid size: ["
              << prop.maxGridSize[0] << ", "
              << prop.maxGridSize[1] << ", "
              << prop.maxGridSize[2] << "]" << std::endl;
    return 0;
}
